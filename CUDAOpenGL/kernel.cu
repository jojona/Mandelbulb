#include "hip/hip_runtime.h"
#include <windows.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include "kernel.h"
//#include "constants.h"

#define PIXELSPERTHREAD 1
#define ACCSECONDARYSQUARE 1

__device__ float EpsilonRaymarch = 0.0005f; // 0.0005f;
__device__ unsigned int MaxRaymarchSteps = 60;
__device__ unsigned int FractalIterations = 10;

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
	}
}

__global__ void kernel(uchar4* pixels, unsigned int width, unsigned int height, glm::mat3 rot, glm::vec3 camPos) {
	int index = blockIdx.x * PIXELSPERTHREAD * blockDim.x + (threadIdx.x * PIXELSPERTHREAD);
	if (index > width*height)
		return;

	for (size_t i = 0; i < PIXELSPERTHREAD; ++i) {
		pixel(pixels, index + i, width, height, rot, camPos);
	}
}

__device__ void pixel(uchar4* pixels, unsigned int index, unsigned int width, unsigned int height, glm::mat3 rotation, glm::vec3 position) {
	const unsigned int x = index % width;
	const unsigned int y = index / width;

	glm::vec3 direction(x - (width / 2.f), y - (height / 2.f), height / 2);
	direction = rotation*direction;
	direction = glm::normalize(direction);

	float distance = 0;

	if (BoundingSphere(direction, position)) {
		distance = RayMarching(position, direction);
	}

	if (distance == 0) {
		//Spectrumbackground(pixels[index], x, y, width, height);
		if (PlaneFloor(direction, position)) {
			pixels[index].w = 0;
			pixels[index].x = 255 & 0xff;
			pixels[index].y = 0 & 0xff;
			pixels[index].z = 0 & 0xff;
		} else {
			pixels[index].w = 0;
			pixels[index].x = 0;
			pixels[index].y = 0;
			pixels[index].z = 0;
		}
	} else {
		distance = 1.f - distance;

		pixels[index].w = 0;
		pixels[index].x = (int)(distance*255.f / MaxRaymarchSteps) & 0xff;
		pixels[index].y = (int)(distance*255.f / MaxRaymarchSteps) & 0xff;
		pixels[index].z = (int)(distance*255.f / MaxRaymarchSteps) & 0xff;
	}
}

/*
 * Colored background.
 */
__device__ void Spectrumbackground(uchar4& pixel, int x, int y, int width, int height) {
	pixel.w = 0;
	pixel.x = (256 * x / (width)) & 0xff;
	pixel.y = (256 * y / (height)) & 0xff;
	pixel.z = 10;
}

/*
 * Ray marching algorithm
 */
__device__ float RayMarching(glm::vec3 pos, glm::vec3 dir) {
	bool hit = false;
	float distance = 0.0f;

	for (int i = 0; i < MaxRaymarchSteps; ++i) {
		//float de = DESphere1(pos);
		//float de = DETetredon(pos);
		//float de = DEMandelbulb1(pos);
		float de = DEMandelbulb2(pos);
		distance += de;
		if (de <= EpsilonRaymarch) {

			return i;
		}
		pos += de * dir;
	}
	return 0.f;
}

/*
 * Distance Estiamtor for a Sphere in origo with radius 3
 */
__device__ float DESphere1(glm::vec3 pos) {
	float distance = glm::length(pos);
	if (distance > 3) {
		distance = distance - 3;
	} else {
		distance = 0;
	}
	return distance;
}

/*
 * Distance estimator for a Tetredon.
 */
__device__ float DETetredon(glm::vec3 z) {
	float Scale = 2.f;

	glm::vec3 a1(1, 1, 1);
	glm::vec3 a2(-1, -1, 1);
	glm::vec3 a3(1, -1, -1);
	glm::vec3 a4(-1, 1, -1);
	glm::vec3 c;
	int n = 0;
	float dist, d;
	while (n < FractalIterations) {
		c = a1; dist = glm::length(z - a1);
		d = glm::length(z - a2); if (d < dist) { c = a2; dist = d; }
		d = glm::length(z - a3); if (d < dist) { c = a3; dist = d; }
		d = glm::length(z - a4); if (d < dist) { c = a4; dist = d; }
		z = Scale*z - c*(Scale - 1.0f);
		n++;
	}

	return glm::length(z) * pow(Scale, float(-n));
}

/*
* Distance estimator for a Mandelbulb. Version 1
*/
__device__ float DEMandelbulb1(glm::vec3 p) {
	glm::vec3 z = p;
	float dr = 1.0f;
	float r = 0.0f;
	for (int i = 0; i < FractalIterations; ++i) {
		r = glm::length(z);
		if (r > EpsilonRaymarch) break;

		// convert to polar coordinates
		float theta = glm::acos(z.z / r);
		float phi = glm::atan(z.y, z.x);
		dr = glm::pow(r, 8.0f - 1.0f)*8.0f*dr + 1.0f;

		// scale and rotate the point
		float zr = glm::pow(r, 8.0f);
		theta = theta*8.f;
		phi = phi*8.f;

		// convert back to cartesian coordinates
		z = zr*glm::vec3(glm::sin(theta)*glm::cos(phi), glm::sin(phi)*glm::sin(theta), glm::cos(theta));
		z += p;
	}
	return 0.5f*glm::log(r)*r / dr;
}

/*
 * Distance estimator for a Mandelbulb. Version 2
 */
__device__ float DEMandelbulb2(glm::vec3 pos) {
	glm::vec3 zz = pos;
	float m = glm::dot(zz, zz);

	float dz = 1.0f;

	for (int i = 0; i < FractalIterations; ++i) {
		float m2 = m*m;
		float m4 = m2*m2;
		dz = 8.0f*glm::sqrt(m4*m2*m)*dz + 1.0f;

		float x = zz.x; float x2 = zz.x*zz.x; float x4 = x2*x2;
		float y = zz.y; float y2 = zz.y* zz.y; float y4 = y2*y2;
		float z = zz.z; float z2 = zz.z*zz.z; float z4 = z2*z2;

		float k3 = x2 + z2;
		float k2 = 1.f / glm::sqrt(k3*k3*k3*k3*k3*k3*k3);
		float k1 = x4 + y4 + z4 - 6.0f*y2*z2 - 6.0f*x2*y2 + 2.0f*z2*x2;
		float k4 = x2 - y2 + z2;

		zz.x = pos.x + 64.0f*x*y*z*(x2 - z2)*k4*(x4 - 6.0f*x2*z2 + z4)*k1*k2;
		zz.y = pos.y + -16.0f*y2*k3*k4*k4 + k1*k1;
		zz.z = pos.z + -8.0f*y*k4*(x4*x4 - 28.0f*x4*x2*z2 + 70.0f*x4*z4 - 28.0f*x2*z2*z4 + z4*z4)*k1*k2;

		m = glm::dot(zz, zz);
		if (m > 1000.0f)
			break;
	}

	return 0.25f*glm::log(m)*glm::sqrt(m) / dz;
}

__device__ bool BoundingSphere(glm::vec3 dir, glm::vec3 pos) {
	return !(glm::length(glm::cross(dir, -pos)) > 1.2f);
}

__device__ bool PlaneFloor(glm::vec3 dir, glm::vec3 pos) {
	float denom = glm::dot(glm::vec3(0, 1, 0), dir);
	if (denom > 0.0001f) // Only visible from above
	{
		float t = glm::dot(glm::vec3(0, 1.1f, 0) - pos, (glm::vec3(0, 1, 0))) / denom;
		if (t >= 0) {
			float distanceFromOrigo = glm::length(pos + t * dir);
			if (distanceFromOrigo > 3 && distanceFromOrigo < 7) {
				return true;
			}
		}
	}
	return false;
}


/*
 * Thread launcher
 */
extern "C" void launch_kernel(uchar4* pos, unsigned int width, unsigned int height, glm::mat3 rot, glm::vec3 campos) {
	// execute the kernel
	int nThreads = 256; // OBS: totalThreads % nThreads = 0
	int totalThreads = height * width / PIXELSPERTHREAD;
	int nBlocks = totalThreads / nThreads;

	kernel<<<nBlocks, nThreads >>>(pos, width, height, rot, campos);

	// Synchronize
	hipDeviceSynchronize();

	checkCUDAError("kernel failed!");
}


extern "C" void launchKernel2(uchar4* pixels, unsigned int width, unsigned int height, glm::mat3 rot, glm::vec3 pos) {
	
	// Change these values if close or far away from the bulb
	if (glm::length(pos) > 5.f) {
		setUp << <1, 1 >> >(.01f, 5, 120);
	} else {
		setUp << <1, 1 >> >(.0005f, 10, 60);
	}
	hipDeviceSynchronize();
	
	// Allocate raymarchSteps and raymarchDistance
	unsigned char* raymarchSteps;
	float * raymarchDistance;

	unsigned int primaryWidth = width % ACCSECONDARYSQUARE == 0 ? width / ACCSECONDARYSQUARE : width / ACCSECONDARYSQUARE + 1;
	unsigned int primaryHeight = height % ACCSECONDARYSQUARE == 0 ? height / ACCSECONDARYSQUARE : height / ACCSECONDARYSQUARE + 1;
	unsigned int primarySize = primaryWidth * primaryHeight;

	hipMalloc((void**)&raymarchSteps, sizeof(unsigned char) * primarySize); // Do only once?
	hipMalloc((void**)&raymarchDistance, sizeof(float) * primarySize); // Do only once?

	int blockThreadsPrimary = 256;
	int totalThreadsPrimary = primarySize;
	int totalBlocksPrimary = totalThreadsPrimary % blockThreadsPrimary == 0 ? totalThreadsPrimary / blockThreadsPrimary : totalThreadsPrimary / blockThreadsPrimary + 1;

	primaryRay<<<totalBlocksPrimary, blockThreadsPrimary >>>(raymarchSteps, raymarchDistance, width, height, primaryWidth, primaryHeight, rot, pos);

	int blockThreads = 256;
	int totalThreads = height * width;
	int totalBlocks = totalThreads % blockThreads == 0 ? totalThreads / blockThreads : totalThreads / blockThreads + 1;

	hipDeviceSynchronize(); // Make sure all primary rays are done

	secondaryRay<<<totalBlocks, blockThreads >>>(pixels, raymarchSteps, raymarchDistance, width, height, primaryWidth, primaryHeight, rot, pos);

	hipDeviceSynchronize(); // Synchronize secondary rays

	hipFree(raymarchSteps); // Do only once?
	hipFree(raymarchDistance); // Do only once?
}


__global__ void primaryRay(unsigned char* raymarchSteps, float* raymarchDistance, unsigned int width, unsigned int height, unsigned int primaryWidth, unsigned int primaryHeight, glm::mat3 rotation, glm::vec3 position) {
	// Calculate pixel index, x, y 
	const unsigned int index = blockIdx.x * blockDim.x + (threadIdx.x);
	if (index >= primaryHeight*primaryWidth) {
		return;
	}
	int squareRadius = ACCSECONDARYSQUARE / 2;

	const unsigned int x = squareRadius + ACCSECONDARYSQUARE * (index % primaryWidth);
	const unsigned int y = squareRadius + ACCSECONDARYSQUARE * (index / primaryWidth);

	glm::vec3 direction(x - (width / 2.f), y - (height / 2.f), height / 2.f);
	direction = rotation*direction;
	direction = glm::normalize(direction);

	glm::vec3 secondDir(x + squareRadius - (width / 2.f), y + squareRadius - (height / 2.f), height / 2.f);
	secondDir = rotation*secondDir;
	secondDir = glm::normalize(secondDir);
	glm::vec3 origin(position);

	float distance = 0;
	int steps = 0;
	// Check bounding sphere
	if (BoundingSphere(direction, position)) {
		// Raymarch as long as all neighbouring rays fit
		//// Only check the corner ray Chapter 4 drive report
		float de = 0.0f; // Maybe create an Estimate or calculation of first circle
		float d = de;
		position += de * direction;

		for (int i = 0; i < MaxRaymarchSteps; ++i) {
			de = DEMandelbulb2(position);
			d += de;

			// Check if all rays are inside here
			if (glm::length(glm::cross(secondDir, position - origin)) > de) {
				de = 0.0f; // TODO change to boolean
			}


			if (de <= EpsilonRaymarch) {
				distance = d;
				steps = i;
				break;
			} 
			position += de * direction;
		}

	}

	// Save result 
	raymarchSteps[index] = steps;
	raymarchDistance[index] = distance;
}

__global__ void secondaryRay(uchar4* pixels, unsigned char* raymarchSteps, float* raymarchDistance, unsigned int width, unsigned int height, unsigned int primaryWidth, unsigned int primaryHeight, glm::mat3 rotation, glm::vec3 position) {
	// Calculate pixel index, x, y
	const unsigned int index = blockIdx.x * blockDim.x + (threadIdx.x);
	if (index >= width * height) {
		return;
	}

	int secondarySteps = 0;

	const unsigned int x = index % width;
	const unsigned int y = index / width;
	const unsigned int primaryIndex = x / ACCSECONDARYSQUARE + (y / ACCSECONDARYSQUARE) * primaryWidth;

	// Calculate start position from primary ray
	glm::vec3 direction(x - (width / 2.f), y - (height / 2.f), height / 2);
	direction = rotation*direction;
	direction = glm::normalize(direction);

	int steps = raymarchSteps[primaryIndex];
	float distance = raymarchDistance[primaryIndex];
	glm::vec3 pos = position + direction * distance;
	bool hit = false;

	if (steps != 0) {
		// Raymarch until eps
		for (int i = steps; i < MaxRaymarchSteps; ++i) {
			secondarySteps++;
			float de = DEMandelbulb2(pos);
			distance += de;
			if (de <= EpsilonRaymarch) {
				hit = true;
				steps = i;
				break;
			}
			pos += de * direction;
		}
	}

	// Draw color to pixels
	if (hit) {
		float color = MaxRaymarchSteps - steps;
		float maxColor = MaxRaymarchSteps;
		//float color = 1.5f + distance - glm::length(position);
		//float maxColor = 1.5f;
		//float color = glm::length(pos) -0.65f;
		//float maxColor = 0.5f;
		
		//float color = MaxRaymarchSteps - (steps * (1 - glm::length(pos) / 1.5f));
		//float maxColor = MaxRaymarchSteps;

		
		glm::vec3 zz = pos;
		float m = glm::dot(zz, zz);

		glm::vec3 orbittrap(abs(zz.x), abs(zz.y), abs(zz.z)); // DO not glm::abs here

		float dz = 1.0f;

		for (int i = 0; i < FractalIterations; ++i) {
			float m2 = m*m;
			float m4 = m2*m2;
			dz = 8.0f*glm::sqrt(m4*m2*m)*dz + 1.0f;

			float x = zz.x; float x2 = zz.x*zz.x; float x4 = x2*x2;
			float y = zz.y; float y2 = zz.y* zz.y; float y4 = y2*y2;
			float z = zz.z; float z2 = zz.z*zz.z; float z4 = z2*z2;

			float k3 = x2 + z2;
			float k2 = 1.f / glm::sqrt(k3*k3*k3*k3*k3*k3*k3);
			float k1 = x4 + y4 + z4 - 6.0f*y2*z2 - 6.0f*x2*y2 + 2.0f*z2*x2;
			float k4 = x2 - y2 + z2;

			zz.x = pos.x + 64.0f*x*y*z*(x2 - z2)*k4*(x4 - 6.0f*x2*z2 + z4)*k1*k2;
			zz.y = pos.y + -16.0f*y2*k3*k4*k4 + k1*k1;
			zz.z = pos.z + -8.0f*y*k4*(x4*x4 - 28.0f*x4*x2*z2 + 70.0f*x4*z4 - 28.0f*x2*z2*z4 + z4*z4)*k1*k2;

			orbittrap = glm::min(orbittrap, glm::vec3(abs(zz.x), abs(zz.y), abs(zz.z))); // Do not glm::abs here

			m = glm::dot(zz, zz);
			if (m > 1000.0f)
				break;

			if (glm::length(glm::vec3(x, y, z)) > 2) {
				dz = i;
				break;
			}
		}
		//color = FractalIterations - dz;
		//maxColor = FractalIterations;
		//printf("%f %f %f\n", orbittrap.x, orbittrap.y, orbittrap.z);

		bool orbitcolor = false; // TODO 

		pixels[index].w = 0;
		if (orbitcolor) {	
			// Orbittrap
			pixels[index].x = (int)(orbittrap.x * 255.f) & 0xff;
			pixels[index].y = (int)(orbittrap.y * 255.f) & 0xff;
			pixels[index].z = (int)(orbittrap.z * 255.f) & 0xff;
		} else {
			pixels[index].x = (int)(color*255.f / maxColor) & 0xff;
			pixels[index].y = (int)(color*255.f / maxColor) & 0xff;
			pixels[index].z = (int)(color*255.f / maxColor) & 0xff;
		}
	} else {
		if (PlaneFloor(direction, position)) {
			pixels[index].w = 0;
			pixels[index].x = 255 & 0xff;
			pixels[index].y = 0 & 0xff;
			pixels[index].z = 0 & 0xff;
		} else {
			pixels[index].w = 0;
			pixels[index].x = 0;
			pixels[index].y = 0;
			pixels[index].z = 0;
		}
	}
}

/*
float time;
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);
setUp<< <1, 1>> >(.0005f, 10, 60);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time, start, stop);
printf("Time %f", time);
*/

__global__ void setUp(float epsilon, unsigned int fractalIterations, unsigned int raymarchsteps) {
	EpsilonRaymarch = epsilon;
	FractalIterations = fractalIterations;
	MaxRaymarchSteps = raymarchsteps;
}